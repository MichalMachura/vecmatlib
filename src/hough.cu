
#include <hip/hip_runtime.h>
/**
 * Computing coordinates of poinnts into Hough space 
 * must be called on length(X) blocks with length(sin_theta) threads per block.
 * 
* int* X, coordinate x of point
* int* Y, coordinate y of point 
* int* OUT, output image of size rho_len x length(sin_theta)
* float* sin_theta, - sinus value calculated for theta vector
* float* cos_theta, - cosinus value calculated for theta
* float rho_resolution, - resolutin of rho
* float rho_len, - number of rows in OUT image 
*
* y = a*x + b
*    |
*    V
* rho = x*sin(theta) + y*cos(theta)
*/
__global__ void hough(	int* X, 
						int* Y, 
						int* OUT, 
						float* sin_theta,
						float* cos_theta,
						float rho_resolution,
						int rho_len)
	{
	// int index = blockIdx.x*blockDim.x+threadIdx.x;
	int theta_idx = threadIdx.x;
	int point_idx = blockIdx.x;
	int theta_len = blockDim.x;
	
	// comute rho value 
	float rho = X[point_idx]*cos_theta[theta_idx] + Y[point_idx]*sin_theta[theta_idx];
	// divide rho by resolution, floor it and add middle value index as position of zero
	int rho_idx = int(rho/rho_resolution) + rho_len/2;
	
	// if rho index is in OUT image range
//	if (rho_idx >= 0 && rho_idx < rho_len)
		// increment point to OUT image
//		atomicAdd(OUT + rho_idx*theta_len + rho_idx, 1);	

    OUT[point_idx*theta_len + theta_idx] = 5;
    }
